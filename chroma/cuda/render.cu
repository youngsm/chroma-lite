#include "hip/hip_runtime.h"
//-*-c-*-

#include "linalg.h"
#include "intersect.h"
#include "mesh.h"
#include "sorting.h"
#include "geometry.h"

#include "stdio.h"

__device__ float4
get_color(const float3 &direction, const Triangle &t, unsigned int rgba)
{
    float3 v01 = t.v1 - t.v0;
    float3 v12 = t.v2 - t.v1;
    
    float3 surface_normal = normalize(cross(v01,v12));

    float cos_theta = dot(surface_normal,-direction);

    if (cos_theta < 0.0f)
	cos_theta = -cos_theta;

    unsigned int a0 = 0xff & (rgba >> 24);
    unsigned int r0 = 0xff & (rgba >> 16);
    unsigned int g0 = 0xff & (rgba >> 8);
    unsigned int b0 = 0xff & rgba;

    float alpha = (255 - a0)/255.0f;

    return make_float4(r0*cos_theta, g0*cos_theta, b0*cos_theta, alpha);
}

extern "C"
{

__global__ void
render(int nthreads, float3 *_origin, float3 *_direction, Geometry *g,
       unsigned int alpha_depth, unsigned int *pixels, float *_dx,
       unsigned int *dxlen, float4 *_color, unsigned int bg_color,
       const float *__restrict__ optix_distances,
       const int *__restrict__ optix_triangles,
       int use_optix)
{
    __shared__ Geometry sg;

    if (threadIdx.x == 0)
	sg = *g;

    __syncthreads();

    int id = blockIdx.x*blockDim.x + threadIdx.x;
	
    if (id >= nthreads)
	return;
    
    g = &sg;

    float3 origin = _origin[id];
    float3 direction = _direction[id];
    unsigned int n = dxlen[id];

    float *dx = _dx + id*alpha_depth;
    float4 *color_a = _color + id*alpha_depth;
    unsigned int n = dxlen[id];

    if( use_optix )
    {
        unsigned int hits = 0;
        float3 norm_dir = normalize(direction);
        for( unsigned int depth = 0; depth < alpha_depth; ++depth )
        {
            float dist = optix_distances ? optix_distances[depth * nthreads + id] : -1.0f;
            int tri_idx = optix_triangles ? optix_triangles[depth * nthreads + id] : -1;
            if( dist < 0.0f || tri_idx < 0 )
                continue;

            Triangle t = get_triangle(g, tri_idx);
            dx[hits]   = dist;
            color_a[hits] = get_color(norm_dir, t, g->colors[tri_idx]);
            ++hits;
            if( hits >= alpha_depth )
                break;
        }

        dxlen[id] = hits;
        if( hits < 1 )
        {
            pixels[id] = bg_color;
            return;
        }
    }
    else
    {
        float distance;
        Node root = get_node(g, 0);

        float3 neg_origin_inv_dir = -origin / direction;
        float3 inv_dir = 1.0f / direction;

        if( n < 1 && !intersect_node( neg_origin_inv_dir, inv_dir, g, root ) )
        {
            pixels[id] = bg_color;
            return;
        }

        unsigned int child_ptr_stack[STACK_SIZE];
        unsigned int nchild_ptr_stack[STACK_SIZE];
        child_ptr_stack[0] = root.child;
        nchild_ptr_stack[0] = root.nchild;

        int curr = 0;

        float3 norm_dir = normalize(direction);

        while( curr >= 0 )
        {
            unsigned int first_child = child_ptr_stack[curr];
            unsigned int nchild      = nchild_ptr_stack[curr];
            curr--;

            for( unsigned int i = first_child; i < first_child + nchild; i++ )
            {
                Node node = get_node( g, i );

                if( intersect_node( neg_origin_inv_dir, inv_dir, g, node ) )
                {
                    if( node.nchild == 0 )
                    {
                        Triangle t = get_triangle( g, node.child );
                        if( intersect_triangle( origin, direction, t, distance ) )
                        {
                            unsigned int insert_index = ( n < 1 ) ? 0 : searchsorted( n, dx, distance );
                            if( insert_index <= alpha_depth - 1 )
                            {
                                insert( alpha_depth, dx, insert_index, distance );
                                unsigned int rgba = g->colors[node.child];
                                float4 color      = get_color( norm_dir, t, rgba );
                                insert( alpha_depth, color_a, insert_index, color );
                                if( n < alpha_depth )
                                    ++n;
                            }
                        }
                    }
                    else
                    {
                        curr++;
                        child_ptr_stack[curr]  = node.child;
                        nchild_ptr_stack[curr] = node.nchild;
                    }
                }
            }
        }

        if( n < 1 )
        {
            pixels[id] = bg_color;
            return;
        }
        dxlen[id] = n;
    }

    n = dxlen[id];

    float scale = 1.0f;
    float fr = 0.0;
    float fg = 0.0;
    float fb = 0.0;
    for (int i=0; i < n; i++) {
	float alpha = color_a[i].w;
	
	fr += scale*color_a[i].x*alpha;
	fg += scale*color_a[i].y*alpha;
	fb += scale*color_a[i].z*alpha;
	
	scale *= (1.0f-alpha);
    }
    float alpha = ((float)((bg_color & 0xFF000000) >> 24))/255.0;
    fr += scale*((float)((bg_color & 0xFF0000) >> 16))*alpha;
    fg += scale*((float)((bg_color & 0xFF00) >> 8))*alpha;
    fb += scale*((float)(bg_color & 0xFF))*alpha;
    scale *= (1.0f-alpha);
    
    unsigned int a;
    if (n < alpha_depth)
	a = floorf(255*(1.0f-scale));
    else
    	a = 255;
    	
    unsigned int red = floorf(fr/(1.0f-scale));
    unsigned int green = floorf(fg/(1.0f-scale));
    unsigned int blue = floorf(fb/(1.0f-scale));

    pixels[id] = a << 24 | red << 16 | green << 8 | blue;
}

} // extern "C"
